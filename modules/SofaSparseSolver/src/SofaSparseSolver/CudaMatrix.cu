/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#ifndef SOFA_COMPONENT_LINEARSOLVER_CUDAMATRIX_H
#define SOFA_COMPONENT_LINEARSOLVER_CUDAMATRIX_H

#include "CudaMatrix.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <ctime>
#include <iostream>

namespace sofa
{

namespace component
{

namespace linearsolver
{
    template<class T>
    struct dp
    {
        T* A, * B;
        int m, n, r;

        dp(T* _A, T* _B, int _m, int _n, int _r) : A(_A), B(_B), m(_m), n(_n), r(_r) {};

        __host__ __device__
            T operator()(size_t idx) {

            T sum = 0.0f;
            int row = idx / r;
            int col = idx - (row * r); // cheaper modulo

            for (int i = 0; i < m; i++)
                sum += A[row * m + i] * B[col * m + i];

            return sum;
        }
    };
    
    void Transpose(double* src, double* dst, unsigned n, unsigned m) {

        // Allocate device memory
        double* d_src;
        double* d_dst;

        // Allocate device memory
        if (hipMalloc(&d_src, sizeof(double) * n * m) != hipSuccess) std::cout << "hipMalloc failed!" << std::endl;
        if (hipMalloc(&d_dst, sizeof(double) * m * n) != hipSuccess) std::cout << "hipMalloc failed!" << std::endl;

        if (hipMemcpy(d_src, src, m * n * sizeof(double), hipMemcpyHostToDevice)) std::cout << "hipMemcpy failed!" << std::endl;
        //hipDeviceSynchronize();

        // cuBLAS handle
        hipblasHandle_t handle;

        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
            std::cout << "CUBLAS initialization failed" << std::endl;

        // Scalaing factors
        double alpha = 1.0;
        double beta = 0.0;

        // Tranpose d_matrix2
        hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, d_src, m, &beta, d_src, n, d_dst, n);
        //hipDeviceSynchronize();

        // Copy back the three matrices
        hipMemcpy(dst, d_dst, sizeof(double) * m * n, hipMemcpyDeviceToHost);
        //hipDeviceSynchronize();

        // Free our memory
        hipFree(d_src);
        hipFree(d_dst);

        hipblasDestroy(handle);
    }

    void MultiplyThrust(double* m1, double* m2, double* result, unsigned m, unsigned n, unsigned r) {

        thrust::device_vector<double> matrix1(m1, m1 + n * m);
        thrust::device_vector<double> matrix2(m2, m2 + m * r);
        thrust::device_vector<double> matrix_result(n * r, 0);

        thrust::transform(thrust::counting_iterator<unsigned>(0),
            thrust::counting_iterator<unsigned>(n * r),
            matrix_result.begin(),
            dp<double>(thrust::raw_pointer_cast(matrix1.data()), thrust::raw_pointer_cast(matrix2.data()), m, n, r));

        hipDeviceSynchronize();

        thrust::copy(matrix_result.begin(), matrix_result.end(), result);
    }

    void MultiplyCUBLAS(double* m1, double* m2, double* result, unsigned m, unsigned n, unsigned r) {

        // Allocate device memory
        double* d_matrix1;
        double* d_matrix2;
        double* d_result;

        // Allocate device memory
        if (hipMalloc(&d_matrix1, sizeof(double) * n * m) != hipSuccess) std::cout << "hipMalloc failed!" << std::endl;
        if (hipMalloc(&d_matrix2, sizeof(double) * m * r) != hipSuccess) std::cout << "hipMalloc failed!" << std::endl;
        if (hipMalloc(&d_result, sizeof(double) * n * r) != hipSuccess) std::cout << "hipMalloc failed!" << std::endl;

        // Copy host to device memory
        if (hipMemcpy(d_matrix1, m1, n * m * sizeof(double), hipMemcpyHostToDevice)) std::cout << "hipMemcpy failed!" << std::endl;
        if (hipMemcpy(d_matrix2, m2, m * r * sizeof(double), hipMemcpyHostToDevice)) std::cout << "hipMemcpy failed!" << std::endl;
        //hipDeviceSynchronize();

        // cuBLAS handle
        hipblasHandle_t handle;

        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
            std::cout << "CUBLAS initialization failed" << std::endl;

        // Scalaing factors
        double alpha = 1.0;
        double beta = 0.0;

        // Calculate: c = (alpha*a) * b + (beta*c)
        // nxr = nxm * mxr
        // Signature: handle, operation, operation, n, r, m, alpha, A, lda, B, ldb,
        // beta, C, ldc    
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, r, m, &alpha, d_matrix2, n, d_matrix1, m, &beta, d_result, r);
        //hipDeviceSynchronize();

        // Copy back the three matrices
        hipMemcpy(result, d_result, sizeof(double) * n * r, hipMemcpyDeviceToHost);
        //hipDeviceSynchronize();

        // Free our memory
        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_result);

        hipblasDestroy(handle);
    }

} // namespace linearsolver

} // namespace component

} // namespace sofa

#endif



